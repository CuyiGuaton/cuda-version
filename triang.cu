#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "consts.h"

//Calcula la distancia entre los puntos |x - y|
__device__ double dist(double x0, double y0, double x1, double y1)
{
	return sqrt(pow(x0 - x1, 2.0) + pow(y0 - y1, 2.0));
}

__device__ int Equality(double a, double b, double epsilon)
{
  return fabs(a - b) < epsilon;
}

//verifica si a es mayor a b de acuerdo a un epsion
__device__ int GreaterEqualthan(double a, double b, double epsilon){
	return Equality(a,b,epsilon) || a > b;
}

/* max_edge_index
 * 
 * Retorna el índice k de la arista máxima de un triángulo i, 
 * descrito por los puntos p0p1p2. Será 0 si p0p1 es máxima.
 * Será 1 si p1p2 lo es. Será 2 si p2p0 lo es.
 * */
__device__ int max_edge_index(int i, double *r, int *p){
     double l0;
     double l1;
     double l2;
     
     int p0;
     int p1;
     int p2;
     
     p0 = p[3*i + 0];
     p1 = p[3*i + 1];
     p2 = p[3*i + 2];
     
     l0 = dist(r[2*p0 + 0], r[2*p0 + 1], r[2*p1 + 0], r[2*p1 + 1]);
     l1 = dist(r[2*p1 + 0], r[2*p1 + 1], r[2*p2 + 0], r[2*p2 + 1]);
     l2 = dist(r[2*p2 + 0], r[2*p2 + 1], r[2*p0 + 0], r[2*p0 + 1]);

     double epsion = 0.001f;
 
     if( (GreaterEqualthan(l0,l1,epsion) && GreaterEqualthan(l1,l2,epsion)) || ( GreaterEqualthan(l0,l2,epsion) && GreaterEqualthan(l2,l1,epsion)))
     {
         return 0;
     }
     else if((GreaterEqualthan(l1,l0,epsion) && GreaterEqualthan(l0,l2,epsion)) || ( GreaterEqualthan(l1,l2,epsion) && GreaterEqualthan(l2,l0,epsion)))
     {
         return 1;
     }
     else
     {
         return 2;
     }
} 


/* same_edge
 * 
 * Indica para las aristas {u,v} y {w,x} si son iguales o no.
 * */
 
 __device__ int same_edge(int u, int v, int w, int x)
 {
     return (u == w && v == x) || (u == x && v == w);
 }


/* get_edge_index
 * 
 * Entrega el índice de la arista {u,v} respecto del triángulo i.
 * */

 __device__  int get_edge_index(int u, int v, int i, int *p)
 {
     int p0;
     int p1;
     int p2;
     
     p0 = p[3*i + 0];
     p1 = p[3*i + 1];
     p2 = p[3*i + 2];
     
     if(same_edge(u, v, p0, p1))
     {
         return 0;
     }
     else if(same_edge(u, v, p1, p2))
     {
         return 1;
     }
     else if(same_edge(u, v, p2, p0))
     {
         return 2;
     }/*
     else
     {
         fprintf(stderr, "%s:%d:%s() ** ERROR ** get_edge_index: Arista {%d,%d} no pertenece al triángulo %d.\n", __FILE__,  __LINE__, __func__, u, v, i);
         exit(EXIT_FAILURE);
     }*/
 }

/* is_nomax_nomax
 * 
 * Indica si la arista compartida entre los triángulos i y j
 * es nomáx-nomáx.
 * */

 __device__ int is_nomax_nomax(int i, int j, int *p, int *max)
 {
     int p0i;
     int p1i;
     int p2i;
     int p0j;
     int p1j;
     int p2j;
     
     p0i = p[3*i + 0];
     p1i = p[3*i + 1];
     p2i = p[3*i + 2];
     
     p0j = p[3*j + 0];
     p1j = p[3*j + 1];
     p2j = p[3*j + 2];
     
     int ij;
     int ii;
     
     if(same_edge(p0i, p1i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p0j, p1j))
     {
         ij = get_edge_index(p0j, p1j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p1j, p2j))
     {
         ij = get_edge_index(p1j, p2j, j, p);
         ii = 2;
     }
     else if(same_edge(p0i, p1i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 0;
     }
     else if(same_edge(p1i, p2i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 1;
     }
     else if(same_edge(p2i, p0i, p2j, p0j))
     {
         ij = get_edge_index(p2j, p0j, j, p);
         ii = 2;
     }
     /*
     else
     {
         fprintf(stderr, "** ERROR ** is_nomax_nomax: Problema insperado para triángulos %d y %d.\n", i, j);
         exit(EXIT_FAILURE);
     }*/
     
     return (ij != max[j]) && (ii != max[i]);
 }

 /* is_max_max
 * 
 * Indica si la arista compartida entre los triángulos i y j
 * es máx-máx.
 * */

 __device__ int is_max_max(int i, int j, int *p, int *max)
{
	int p0i;
	int p1i;
	int p2i;
	
	int p0j;
	int p1j;
	int p2j;
	
	p0i = p[3*i + 0];
	p1i = p[3*i + 1];
	p2i = p[3*i + 2];
	
	p0j = p[3*j + 0];
	p1j = p[3*j + 1];
	p2j = p[3*j + 2];
	
	int ij;
	int ii;
	
	if(same_edge(p0i, p1i, p0j, p1j))
	{
		ij = get_edge_index(p0j, p1j, j, p);
		ii = 0;
	}
	else if(same_edge(p1i, p2i, p0j, p1j))
	{
		ij = get_edge_index(p0j, p1j, j, p);
		ii = 1;
	}
	else if(same_edge(p2i, p0i, p0j, p1j))
	{
		ij = get_edge_index(p0j, p1j, j, p);
		ii = 2;
	}
	else if(same_edge(p0i, p1i, p1j, p2j))
	{
		ij = get_edge_index(p1j, p2j, j, p);
		ii = 0;
	}
	else if(same_edge(p1i, p2i, p1j, p2j))
	{
		ij = get_edge_index(p1j, p2j, j, p);
		ii = 1;
	}
	else if(same_edge(p2i, p0i, p1j, p2j))
	{
		ij = get_edge_index(p1j, p2j, j, p);
		ii = 2;
	}
	else if(same_edge(p0i, p1i, p2j, p0j))
	{
		ij = get_edge_index(p2j, p0j, j, p);
		ii = 0;
	}
	else if(same_edge(p1i, p2i, p2j, p0j))
	{
		ij = get_edge_index(p2j, p0j, j, p);
		ii = 1;
	}
	else if(same_edge(p2i, p0i, p2j, p0j))
	{
		ij = get_edge_index(p2j, p0j, j, p);
		ii = 2;
	}
	else
	{
	//	fprintf(stderr, "** ERROR ** is_max_max: Problema insperado para triángulos %d y %d.\n", i, j);
    //exit(EXIT_FAILURE);
	}
	
	return (ij == max[j]) && (ii == max[i]);
}




/* edge_belongs_to
 * 
 * Indica si arista {k,l} pertenece al triángulo i.
 * */

 __device__ int edge_belongs_to(int k, int l, int i, int *p)
 {
     return same_edge(k, l, p[3*i + 0], p[3*i + 1])
                     || same_edge(k, l, p[3*i + 1], p[3*i + 2])
                     || same_edge(k, l, p[3*i + 2], p[3*i + 0]);
 }

__global__ void label_longest_edges(int *cu_max, double *cu_r, int *cu_triangles)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    cu_max[i] = max_edge_index(i,cu_r, cu_triangles);
}

__global__ void label_frontier_edges(int *cu_max, int *cu_triangles, int *cu_adj, int *cu_seed)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    for(int j = 0; j < 3; j++)
    {
        if(cu_adj[3*i +j] < 0 || is_nomax_nomax(i, cu_adj[3*i + j], cu_triangles, cu_max))
            cu_adj[3*i + j] = NO_ADJ;
        

        if(cu_adj[3*i +j] >= 0 && is_max_max(i, cu_adj[3*i + j], cu_triangles, cu_max) && cu_seed[cu_adj[3*i + j]] == FALSE)
            cu_seed[i] = TRUE;
        
    }   
}

