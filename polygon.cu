#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "consts.h"



__device__ int count_FrontierEdges(int triangle, int *cu_adj){
    int adj_counter = 0;
    int j;
    for(j = 0; j < 3; j++){ 
        if(cu_adj[3*triangle + j] == NO_ADJ){
            adj_counter++;
        }
    }
    return adj_counter;
}


__device__ int generate_polygon(int * poly, int * triangles, int * adj, double *r, int i) {
    int ind_poly = 0;
	
	int initial_point = 0;
	int end_point = 0;
	
	int t0;
	int t1;	
	int t2;
    int ind0;
    int ind1;
    int ind2;
	int continuous;
	int k, j, aux;
	int origen;

    int num_FrontierEdges = count_FrontierEdges(i, adj);
    //debug_print("Generando polinomio con triangulo %d FE %d\n", i, num_FrontierEdges);
    /*si tiene 3 se agregan y se corta el ciclo*/
    if (num_FrontierEdges == 3) {
        //debug_print("T %d Tiene 3 Frontier edge, se guardan así\n", i);
        poly[ind_poly] = triangles[3 * i + 0];
        ind_poly++;
        poly[ind_poly] = triangles[3 * i + 1];
        ind_poly++;
        poly[ind_poly] = triangles[3 * i + 2];
        ind_poly++;

        //visited[i] = TRUE;
        return ind_poly;
    } else if(num_FrontierEdges == 2) {
        //debug_print("T %d Tiene 2 Frontier edge, es oreja, se usa como semilla para generar el poly\n", i);
        /*si tiene dos FE se agregan y se empieza el ciclo*/
        for(j = 0; j<3; j++){
            ind0 = 3*i + j;
            ind1 = 3*i + (j+1)%3;
            ind2 = 3*i + (j+2)%3;
            if(adj[ind0] == NO_ADJ && adj[ind1] == NO_ADJ){
                poly[ind_poly] = triangles[ind1];
                ind_poly++;
                poly[ind_poly] = triangles[ind2];
                ind_poly++;

                initial_point = triangles[ind1];
                end_point = triangles[ind0];  
            }
        }
    }else if (num_FrontierEdges == 1){
        //debug_print("T %d Tiene 1 Frontier edge,se usa como FE initial\n", i);
        /*si tiene dos FE se agregan y se empieza el ciclo*/
        for(j = 0; j<3; j++){
            if(adj[3*i + j] == NO_ADJ){
                poly[ind_poly] = triangles[3*i + (j+1)%3];
                ind_poly++;
                initial_point = triangles[3*i + (j+1)%3];

                end_point = triangles[3*i + (j+2)%3];  
            }
        }
    }else {
        end_point = triangles[3*i + 0];
        initial_point = triangles[3*i + 0];
    }
    
    
    /*se marca como visitado */
    //visited[i] = TRUE;
    num_FrontierEdges = 0;
    k = i;
    aux = k;
    k = get_adjacent_triangle_share_endpoint(k, k, end_point, triangles, adj); /* cambia el indice */
    continuous = is_continuous(k, end_point, triangles);
    origen = aux;
//        //debug_print("k %d origen %d, conti %d\n", k, origen, continuous);
    //debug_print("T_inicial %d | Triangles %d %d %d | ADJ  %d %d %d\n", i, triangles[3*i + 0], triangles[3*i + 1], triangles[3*i + 2], adj[3*i + 0], adj[3*i + 1], adj[3*i + 2]);
    //debug_print("initial_point %d endpoint %d | T_sig %d\n", initial_point, end_point, k);

    int triangugulo_initial = i;
    while (initial_point != end_point || triangugulo_initial != k) {

        /*se marca el triangulo visto como visitado y se suma al area del poligono */
        
      //  visited[k] = TRUE;
        t0 = adj[3 * k + 0];
        t1 = adj[3 * k + 1];
        t2 = adj[3 * k + 2];

        num_FrontierEdges = count_FrontierEdges(k, adj);
        //debug_print("FE %d | origen %d t %d | Triangles %d %d %d | ADJ  %d %d %d\n", num_FrontierEdges, origen, k, triangles[3*k + 0], triangles[3*k + 1], triangles[3*k + 2], adj[3*k + 0], adj[3*k + 1], adj[3*k + 2]);
        if(origen == -2)
            exit(0);
        if (num_FrontierEdges == 2 && continuous != -1) {
            /* ///////////////////si tiene 2 frontier edge se agregan a poly //////////////////////////////////// */

            if (t0 == NO_ADJ && t1 == NO_ADJ) {
                /*si endpoint es continua a t0  y t0-t1 son fe*/
                if (continuous == 1) {
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;

                    end_point = triangles[3 * k + 0];

                } else if (continuous == 0) {
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;

                    end_point = triangles[3 * k + 1];
                }
            } else if (t2 == NO_ADJ && t0 == NO_ADJ) {
                /*si endpoint es continua a t2  y t2-t0 son fe*/
                if (continuous == 0) {
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;

                    end_point = triangles[3 * k + 2];

                } else if (continuous == 2) {
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;

                    end_point = triangles[3 * k + 0];

                }
            } else if (t1 == NO_ADJ && t2 == NO_ADJ) {
                /*si endpoint es continua a t1 y t1-t2 son fe*/
                if (continuous == 2) {
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;

                    end_point = triangles[3 * k + 1];

                } else if (continuous == 1) {
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;

                    end_point = triangles[3 * k + 2];

                }
            } else {
                //fprintf(stderr, "** ERROR ** Adding 2 fronter edges\n");
                //fprintf(stderr, "** ERROR ** k %d t %d %d %d ini %d end %d \n", k, t0, t1, t2, initial_point, end_point);
            }

            aux = k;
            k = get_adjacent_triangle_share_endpoint(k, -1, end_point, triangles, adj); /* se le permite volver al triangulo anterior */
            continuous = is_continuous(k, end_point, triangles);
            origen = aux;

        } else if (num_FrontierEdges == 1 && continuous != -1) {
            /* ///////////////////si solo se tiene 1 frontier edge //////////////////////////////////// */
            if (t0 == NO_ADJ) {
                /*si endpoint es continua a t0  y t0 es fe*/
                if (continuous == 1) {
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;

                    end_point = triangles[3 * k + 2];

                } else if (continuous == 2) {
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;

                    end_point = triangles[3 * k + 1];

                }
            } else if (t2 == NO_ADJ) {
                /*si endpoint es continua a t2  y t2 es fe*/
                if (continuous == 0) {
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;

                    end_point = triangles[3 * k + 1];

                } else if (continuous == 1) {
                    poly[ind_poly] = triangles[3 * k + 1];
                    ind_poly++;

                    end_point = triangles[3 * k + 0];

                }
            } else if (t1 == NO_ADJ) {
                /*si endpoint es continua a t1  y t1 es fe*/
                if (continuous == 2) {
                    poly[ind_poly] = triangles[3 * k + 2];
                    ind_poly++;

                    end_point = triangles[3 * k + 0];

                } else if (continuous == 0) {
                    poly[ind_poly] = triangles[3 * k + 0];
                    ind_poly++;

                    end_point = triangles[3 * k + 2];

                }
            } else {
                //fprintf(stderr, "** ERROR ** Adding 1 fronter edges\n");
                //fprintf(stderr, "** ERROR ** k %d t %d %d %d ini %d end %d conti %d\n", k, t0, t1, t2, initial_point, end_point, continuous);
            }
            /*si es continuo y tiene 1 fe no puede volver, ind si se guarda  o no*/
            aux = k;
            k = get_adjacent_triangle_share_endpoint(k, origen, end_point, triangles, adj); /* cambia el indice */
            continuous = is_continuous(k, end_point, triangles);
            origen = aux;
        } else {
            /*si no es continuo no puede regresar de donde venía*/
            aux = k;
            k = get_adjacent_triangle_share_endpoint(k, origen, end_point, triangles, adj); /* cambia el indice */
            continuous = is_continuous(k, end_point, triangles);
            origen = aux;
        }

    }
    
    return ind_poly;
}


__global__ void generate_mesh(int *cu_triangles, int *cu_adj, double *cu_r, int *cu_seed){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(cu_seed[i] == TRUE){
        int poly[1000];

        int length_poly = generate_polygon(poly, cu_triangles, cu_adj, cu_r, i);
        int num_BE = count_BarrierEdges(poly, length_poly);
    }
    
}