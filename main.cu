#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>   
#include <stdlib.h>     /* exit, EXIT_FAILURE */
#include "detri2.h"
#include "polymesh.h"
#include <vector> 
#include <chrono>
#include <iomanip>
#include <cstdlib>




#include "io.h"
#include "consts.h"


//cuda
#include "triangle.cuh"
#include "polygon.cuh"


#ifdef DEBUG
#define DEBUG_TEST 1
#else
#define DEBUG_TEST 0
#endif

#define debug_block(fmt) do { if (DEBUG_TEST){ fmt }} while (0)
#define debug_print(fmt, ...) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__, __LINE__, __func__, __VA_ARGS__); } while (0)
#define debug_msg(fmt) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__,  __LINE__, __func__); } while (0)


int main(int argc, char* argv[]){


    int nparam = 3;
    //char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("test.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506randompoints.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506equilateral.node")};
    int print_triangles = 0;
    char* ppath;
    //char* ppath = const_cast<char*> ("test");
    //TMesh *Tr = new TMesh(nparam, params);    
	auto tb_delaunay = std::chrono::high_resolution_clock::now();
	TMesh *Tr = new TMesh(argc, argv);    
	
	auto te_delaunay = std::chrono::high_resolution_clock::now();
    //Tr->print();
    
	int tnumber, pnumber, i,j;
	double *r;
	int *triangles;
	int *adj;
    int *seed;
	int *max;
	int *mesh;
	int *disconnect;

	int i_mesh = 0;	
	int length_poly = 0;
	int id_pos_poly = 0;
	
    tnumber = Tr->tnumber;
    pnumber = Tr->pnumber;

	max = (int *)malloc(tnumber*sizeof(int));
	disconnect = (int *)malloc(3*tnumber*sizeof(int));
	seed = (int *)malloc(tnumber*sizeof(int));
    r = (double *)malloc(2*tnumber*sizeof(double));
    adj =(int *)malloc(3*tnumber*sizeof(int));
    triangles = (int *)malloc(3*tnumber*sizeof(int));
	mesh = (int *)malloc(3*tnumber*sizeof(int));
	

	//Cuda functions
    // Initialize device pointers.
    double *cu_r;
	int *cu_triangles;
	int *cu_adj;
    int *cu_seed;
	int *cu_max;
	int *cu_disconnect;
	int *cu_mesh;

	// Allocate device memory.
	hipMalloc((void**) &cu_max, tnumber*sizeof(int));
	hipMalloc((void**) &cu_seed, tnumber*sizeof(int));
	hipMalloc((void**) &cu_disconnect, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_r, 2*tnumber*sizeof(double));
	hipMalloc((void**) &cu_triangles, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_adj, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_mesh, 3*tnumber*sizeof(int));


	/* Llamada a detr2 */

    int idx =0;
    //copiar arreglo de vertices
    //std::cout<<"pnumber "<<pnumber<<std::endl;
    for (i = 0; i < Tr->trimesh->ct_in_vrts; i++) {
        if (!Tr->trimesh->io_keep_unused) { // no -IJ
            if (Tr->trimesh->in_vrts[i].typ == UNUSEDVERTEX) continue;
        }
        r[2*i + 0]= Tr->trimesh->in_vrts[i].crd[0];
        r[2*i + 1]= Tr->trimesh->in_vrts[i].crd[1];
        //std::cout<<idx<<" ("<<r[2*i + 0]<<", "<<r[2*i + 1]<<") "<<std::endl;
        Tr->trimesh->in_vrts[i].idx = idx;
        idx++;
    }
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++) {
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted()) continue;
        if (tri->is_hulltri()) {
            tri->idx = -1;
        } else {
            tri->idx = idx;
            idx++;
        }
    }

    //std::cout<<"tnumber: "<<Tr->trimesh->tr_tris->objects - Tr->trimesh->ct_hullsize<<std::endl;
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++)
    {
        
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted() || tri->is_hulltri()) continue;
        triangles[3*idx+0] = tri->vrt[0]->idx;
        triangles[3*idx+1] = tri->vrt[1]->idx;
        triangles[3*idx+2] = tri->vrt[2]->idx;
        adj[3*idx+ 0] = tri->nei[0].tri->idx;
        adj[3*idx+ 1] = tri->nei[1].tri->idx;
        adj[3*idx+ 2] = tri->nei[2].tri->idx;
        //std::cout<<idx<<" | "<<triangles[3*idx+0]<<" "<<triangles[3*idx+1]<<" "<<triangles[3*idx+2]<<" | ";
        //std::cout<<adj[3*idx+ 0]<<" "<<adj[3*idx+ 1]<<" "<<adj[3*idx+ 2]<<" | "<<std::endl;
        idx++;
    }
	delete Tr;

	for(i = 0; i <tnumber; i++)
		seed[i] = TRUE;

    // Transfer arrays a and b to device.
    hipMemcpy(cu_r, r,                 2*tnumber*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cu_triangles, triangles, 3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_adj, adj,             3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_seed, seed,    		tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_max, max,             tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_disconnect, disconnect,             3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_mesh, mesh,           3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	
	
	//Algoritmo de testeo para ver si visitan todos los triangulos
	test_kernel<<<tnumber, 1>>>(cu_seed, tnumber);
	hipMemcpy(seed, cu_seed, tnumber*sizeof(int), hipMemcpyDeviceToHost);
	for (i = 0; i < tnumber; i++){
		if(seed[i] == TRUE)
			return 0;
	}
	
	
	//Label phase
	//__global__ void label_longest_edges(int *cu_max, double *cu_r, int *cu_triangles, int tnumber);
	label_longest_edges<<<tnumber, 1>>>(cu_max, cu_r, cu_triangles, tnumber);
	hipDeviceSynchronize();
	//__global__ void label_frontier_edges(int *cu_max, int *disconnect, int *cu_triangles, int *cu_adj, int tnumber);
	label_frontier_edges<<<tnumber, 1>>>(cu_max, cu_disconnect, cu_triangles, cu_adj, tnumber);
	hipDeviceSynchronize();
	//__global__ void disconnect_edges(int *cu_adj, int* cu_disconnect, inttnumber)
	disconnect_edges<<<tnumber, 1>>>(cu_adj, cu_disconnect, tnumber);
	hipDeviceSynchronize();
	get_seeds<<<tnumber, 1>>>(cu_max, cu_triangles, cu_adj, cu_seed, tnumber);
	hipDeviceSynchronize();

	
	
	__global__ void get_seeds(int *cu_max, int *cu_triangles, int *cu_adj, int *cu_seed, int tnumber);
	
	__global__ void test_kernel(int *cu_seed, int tnumber);

	hipMemcpy(seed, cu_seed,tnumber*sizeof(int), hipMemcpyDeviceToHost);
	int regiones = 0;
	for (i = 0; i < tnumber; i++)
	{	
		if(seed[i] == TRUE){
			seed[regiones] = i;
			regiones++;
		}
	}
	for (i = 0; i < regiones; i++)
		std::cout<<seed[i]<<" ";
	std::cout<<"\nregiones = "<<regiones<<std::endl;

	//generate_mesh<<<tnumber, 1>>>(cu_triangles, cu_adj, cu_r, cu_seed,cu_mesh);
	
	free(r);
	free(triangles);
	free(adj);
	free(seed );
	free(mesh);
	free(max);
    
	return EXIT_SUCCESS;
}
    

