#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>   
#include <stdlib.h>     /* exit, EXIT_FAILURE */
#include "detri2.h"
#include "polymesh.h"
#include <vector> 
#include <chrono>
#include <iomanip>
#include <cstdlib>




#include "io.h"
#include "consts.h"


//cuda
#include "triangle.cuh"
#include "polygon.cuh"


#ifdef DEBUG
#define DEBUG_TEST 1
#else
#define DEBUG_TEST 0
#endif

#define debug_block(fmt) do { if (DEBUG_TEST){ fmt }} while (0)
#define debug_print(fmt, ...) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__, __LINE__, __func__, __VA_ARGS__); } while (0)
#define debug_msg(fmt) do { if (DEBUG_TEST) fprintf(stderr, "%s:%d:%s(): " fmt, __FILE__,  __LINE__, __func__); } while (0)


int main(int argc, char* argv[]){


    int nparam = 3;
    //char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("test.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506randompoints.node")};
	//char* params[] = {const_cast<char*> ("./detri2"), const_cast<char*> ("-z"), const_cast<char*> ("506equilateral.node")};
    int print_triangles = 0;
    char* ppath;
    //char* ppath = const_cast<char*> ("test");
    //TMesh *Tr = new TMesh(nparam, params);    
	auto tb_delaunay = std::chrono::high_resolution_clock::now();
	TMesh *Tr = new TMesh(argc, argv);    
	
	auto te_delaunay = std::chrono::high_resolution_clock::now();
    //Tr->print();
    
	int tnumber, pnumber, i,j;
	double *r;
	int *triangles;
	int *adj;
    int *seed;
	int *max;
	int *mesh;
	int *disconnect;

	
	
    tnumber = Tr->tnumber;
    pnumber = Tr->pnumber;

	max = (int *)malloc(tnumber*sizeof(int));
	disconnect = (int *)malloc(3*tnumber*sizeof(int));
	seed = (int *)malloc(tnumber*sizeof(int));
    r = (double *)malloc(2*tnumber*sizeof(double));
    adj =(int *)malloc(3*tnumber*sizeof(int));
    triangles = (int *)malloc(3*tnumber*sizeof(int));
	mesh = (int *)malloc(3*tnumber*sizeof(int));
	

	//Cuda functions
    // Initialize device pointers.
    double *cu_r;
	int *cu_triangles;
	int *cu_adj;
    int *cu_seed;
	int *cu_max;
	int *cu_disconnect;
	int *cu_mesh;

	// Allocate device memory.
	hipMalloc((void**) &cu_max, tnumber*sizeof(int));
	hipMalloc((void**) &cu_seed, tnumber*sizeof(int));
	hipMalloc((void**) &cu_disconnect, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_r, 2*tnumber*sizeof(double));
	hipMalloc((void**) &cu_triangles, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_adj, 3*tnumber*sizeof(int));
	hipMalloc((void**) &cu_mesh, 3*tnumber*sizeof(int));


	/* Llamada a detr2 */

    int idx =0;
    //copiar arreglo de vertices
    //std::cout<<"pnumber "<<pnumber<<std::endl;
    for (i = 0; i < Tr->trimesh->ct_in_vrts; i++) {
        if (!Tr->trimesh->io_keep_unused) { // no -IJ
            if (Tr->trimesh->in_vrts[i].typ == UNUSEDVERTEX) continue;
        }
        r[2*i + 0]= Tr->trimesh->in_vrts[i].crd[0];
        r[2*i + 1]= Tr->trimesh->in_vrts[i].crd[1];
        //std::cout<<idx<<" ("<<r[2*i + 0]<<", "<<r[2*i + 1]<<") "<<std::endl;
        Tr->trimesh->in_vrts[i].idx = idx;
        idx++;
    }
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++) {
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted()) continue;
        if (tri->is_hulltri()) {
            tri->idx = -1;
        } else {
            tri->idx = idx;
            idx++;
        }
    }

    //std::cout<<"tnumber: "<<Tr->trimesh->tr_tris->objects - Tr->trimesh->ct_hullsize<<std::endl;
    idx = 0;
    for (int i = 0; i < Tr->trimesh->tr_tris->used_items; i++)
    {
        
        detri2::Triang* tri = (detri2::Triang *) Tr->trimesh->tr_tris->get(i);
        if (tri->is_deleted() || tri->is_hulltri()) continue;
        triangles[3*idx+0] = tri->vrt[0]->idx;
        triangles[3*idx+1] = tri->vrt[1]->idx;
        triangles[3*idx+2] = tri->vrt[2]->idx;
        adj[3*idx+ 0] = tri->nei[0].tri->idx;
        adj[3*idx+ 1] = tri->nei[1].tri->idx;
        adj[3*idx+ 2] = tri->nei[2].tri->idx;
        //std::cout<<idx<<" | "<<triangles[3*idx+0]<<" "<<triangles[3*idx+1]<<" "<<triangles[3*idx+2]<<" | ";
        //std::cout<<adj[3*idx+ 0]<<" "<<adj[3*idx+ 1]<<" "<<adj[3*idx+ 2]<<" | "<<std::endl;
        idx++;
    }
	delete Tr;

	for(i = 0; i <tnumber; i++){
		seed[i] = FALSE;
		disconnect[3*i+0] = FALSE;
		disconnect[3*i+1] = FALSE;
		disconnect[3*i+2] = FALSE;
	}
		

    // Transfer arrays to device.
    hipMemcpy(cu_r, r,                   2*tnumber*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(cu_triangles, triangles,   3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_adj, adj,               3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_seed, seed,    		  tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_max, max,               tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_disconnect, disconnect, 3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cu_mesh, mesh,             3*tnumber*sizeof(int), hipMemcpyHostToDevice);
	
	
	//Algoritmo de testeo para ver si visitan todos los triangulos
	test_kernel<<<tnumber, 1>>>(cu_seed, tnumber);
	hipMemcpy(seed, cu_seed, tnumber*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	for (i = 0; i < tnumber; i++){
		if(seed[i] == TRUE)
			return 0;
	}
	
	
	//Label phase
	//Etiquetar el más largo;
	label_longest_edges<<<tnumber, 1>>>(cu_max, cu_r, cu_triangles, tnumber);
	hipDeviceSynchronize();

	//Encontrar un triangulo semilla asociado al arco terminal
	get_seeds<<<tnumber, 1>>>(cu_max, cu_triangles, cu_adj, cu_seed, tnumber);
	hipDeviceSynchronize();

	//Etiquetar label frontier-edges
	label_frontier_edges<<<tnumber, 1>>>(cu_max, cu_disconnect, cu_triangles, cu_adj, tnumber);
	hipDeviceSynchronize();
	
	//Desconectar frontier-edges
	disconnect_edges<<<tnumber, 1>>>(cu_adj, cu_disconnect, tnumber);
	hipDeviceSynchronize();


	//hipMemcpy(adj, cu_adj,3*tnumber*sizeof(int), hipMemcpyDeviceToHost);
	//for (i = 0; i < tnumber; i++)
	//	std::cout<<adj[3*i+0]<<" "<<adj[3*i+1]<<" "<<adj[3*i+2]<<"\n";

	//Se ordenan las semillas
	hipMemcpy(seed, cu_seed,tnumber*sizeof(int), hipMemcpyDeviceToHost);
	int num_region = 0;
	for (i = 0; i < tnumber; i++)
	{	
		if(seed[i] == TRUE){
			seed[num_region] = i;
			num_region++;
		}
	}
	for (i = 0; i < num_region; i++)
		std::cout<<seed[i]<<" ";
	std::cout<<"\nregiones = "<<num_region<<std::endl;

	//se consigue el indice de la malla i_mesh
	int i_mesh = 0;
	int *cu_i_mesh;
	hipMalloc((void**) &cu_i_mesh, sizeof(int));
	hipMemcpy(cu_i_mesh, &i_mesh, 1*sizeof(int), hipMemcpyHostToDevice);
	
	generate_mesh<<<num_region, 1>>>(cu_triangles, cu_adj, cu_r, cu_seed, cu_mesh, num_region, cu_i_mesh);
	hipMemcpy(&i_mesh, cu_i_mesh,sizeof(int), hipMemcpyDeviceToHost);
	std::cout<<"\ni_mesh = "<<i_mesh<<std::endl;

	hipMemcpy(mesh, cu_mesh,3*tnumber*sizeof(int), hipMemcpyDeviceToHost);
	//std::cout<<"mesh[i] = ";
	//for (i = 0; i < num_region; i++)
	//	std::cout<<mesh[i]<<" ";
	//std::cout<<"\n";

	i = 0;
	while(i < i_mesh){
		int length_poly = mesh[i];
		i++;
		std::cout<<length_poly<<": ";
		for(j=0; j < length_poly;j++){
			std::cout<<mesh[i] <<" ";
			i++;
		}
		std::cout<<"\n";
	}

	free(r);
	free(triangles);
	free(adj);
	free(seed );
	free(mesh);
	free(max);
    
	return EXIT_SUCCESS;
}
    

